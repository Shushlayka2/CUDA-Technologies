#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <fstream>
#include <malloc.h>
#include <vector>

using namespace std;

#define BLOCK_SIZE 128
#define ArraySize 100000
#define AbsMaxVal 10

void generate_random_arr(float* A);
void exec_first_condition(float* A, float* B);
void exec_second_condition(float* A, float* B);
void exec_third_condition(float* A, float* B);
float sum_particles_host(float* d_A_even, float* d_A_odd);

texture<float, 1, hipReadModeElementType> FirstArrElementsRef;
texture<float, 1, hipReadModeElementType> SecondArrElementsRef;
texture<float, 1, hipReadModeElementType> SumArrElementsRef;

__global__ void mult_particles_first(float* C)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < ArraySize)
        C[i] = __fmul_rn(tex1D(FirstArrElementsRef, i), tex1D(SecondArrElementsRef, i));
}

__global__ void mult_particles_second(float* C)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < ArraySize)
        C[i] = __fmul_rn(tex1Dfetch(FirstArrElementsRef, i), tex1D(SecondArrElementsRef, i));
}

__global__ void mult_particles_third(float* C)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < ArraySize)
        C[i] = __fmul_rn(tex1Dfetch(FirstArrElementsRef, i), tex1Dfetch(SecondArrElementsRef, i));
}

__global__ void sum_particles(float* A, int size, int iterations)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < iterations)
    {
        int start = idx * BLOCK_SIZE;
        int end = min((idx + 1) * BLOCK_SIZE, size);
        A[idx] = 0;
        for (int j = start; j < end; j++)
            A[idx] = __fadd_rn(A[idx], tex1Dfetch(SumArrElementsRef, j));
    }
}

int main()
{
    /*ifstream file;
    file.open("input.txt");
    vector<float> data;
    int length = 0;
    while (!file.eof())
    {
        float a;
        file >> a;
        data.push_back(a);
        length++;
    }
    file.close();
    length /= 2;
    float* A, * B;
    A = &data[0];
    B = &data[length];*/

    float* A, * B;
    A = (float*)malloc(sizeof(float) * ArraySize);
    B = (float*)malloc(sizeof(float) * ArraySize);
    generate_random_arr(A);
    generate_random_arr(B);

    exec_first_condition(A, B);
    exec_second_condition(A, B);
    exec_third_condition(A, B);
    
    //data.clear();
    hipUnbindTexture(FirstArrElementsRef);
    hipUnbindTexture(SecondArrElementsRef);
    hipUnbindTexture(SumArrElementsRef);
	return 0;
}

void exec_first_condition(float* A, float* B)
{
    float* d_C_odd, *d_C_even;
    hipArray* d_A, * d_B;
    size_t size = sizeof(float) * ArraySize;
    int GRID_SIZE = ArraySize / BLOCK_SIZE + (ArraySize % BLOCK_SIZE != 0 ? 1 : 0);
    hipMalloc((void**)&d_C_odd, size);
    hipMalloc((void**)&d_C_even, size);
    hipMallocArray(&d_A, &FirstArrElementsRef.channelDesc, ArraySize, 1);
    hipMallocArray(&d_B, &SecondArrElementsRef.channelDesc, ArraySize, 1);
    hipMemcpyToArray(d_A, 0, 0, A, size, hipMemcpyHostToDevice);
    hipMemcpyToArray(d_B, 0, 0, B, size, hipMemcpyHostToDevice);
    hipBindTextureToArray(FirstArrElementsRef, d_A);
    hipBindTextureToArray(SecondArrElementsRef, d_B);

    float KernelTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    mult_particles_first << <GRID_SIZE, BLOCK_SIZE >> > (d_C_odd);
    hipBindTexture(0, SumArrElementsRef, d_C_odd, size);
    float result = sum_particles_host(d_C_even, d_C_odd);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&KernelTime, start, stop);

    printf("First condition:\n");
    printf("Result: %f\n", result);
    printf("Elapsed time: %f\n", KernelTime);
    printf("\n");

    hipUnbindTexture(SumArrElementsRef);
    hipFree(d_C_odd);
    hipFree(d_C_even);
    hipFreeArray(d_A);
    hipFreeArray(d_B);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void exec_second_condition(float* A, float* B)
{
    float* d_A, *d_C_odd, *d_C_even;
    hipArray* d_B;
    size_t size = sizeof(float) * ArraySize;
    int GRID_SIZE = ArraySize / BLOCK_SIZE + (ArraySize % BLOCK_SIZE != 0 ? 1 : 0);
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_C_odd, size);
    hipMalloc((void**)&d_C_even, size);
    hipMallocArray(&d_B, &SecondArrElementsRef.channelDesc, ArraySize, 1);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipBindTexture(0, FirstArrElementsRef, d_A, size);
    hipMemcpyToArray(d_B, 0, 0, B, size, hipMemcpyHostToDevice);
    hipBindTextureToArray(SecondArrElementsRef, d_B);

    float KernelTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    mult_particles_second << <GRID_SIZE, BLOCK_SIZE >> > (d_C_odd);
    hipBindTexture(0, SumArrElementsRef, d_C_odd, size);
    float result = sum_particles_host(d_C_even, d_C_odd);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&KernelTime, start, stop);

    printf("Second condition:\n");
    printf("Result: %f\n", result);
    printf("Elapsed time: %f\n", KernelTime);
    printf("\n");

    hipUnbindTexture(SumArrElementsRef);
    hipFree(d_C_odd);
    hipFree(d_C_even);
    hipFree(d_A);
    hipFreeArray(d_B);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void exec_third_condition(float* A, float* B)
{
    float* d_A, *d_B, * d_C_odd, * d_C_even;
    size_t size = sizeof(float) * ArraySize;
    int GRID_SIZE = ArraySize / BLOCK_SIZE + (ArraySize % BLOCK_SIZE != 0 ? 1 : 0);
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C_odd, size);
    hipMalloc((void**)&d_C_even, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    hipBindTexture(0, FirstArrElementsRef, d_A, size);
    hipBindTexture(0, SecondArrElementsRef, d_B, size);

    float KernelTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    mult_particles_third << <GRID_SIZE, BLOCK_SIZE >> > (d_C_odd);
    hipBindTexture(0, SumArrElementsRef, d_C_odd, size);
    float result = sum_particles_host(d_C_even, d_C_odd);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&KernelTime, start, stop);

    printf("Third condition:\n");
    printf("Result: %f\n", result);
    printf("Elapsed time: %f\n", KernelTime);
    printf("\n");

    hipUnbindTexture(SumArrElementsRef);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C_odd);
    hipFree(d_C_even);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

float sum_particles_host(float* d_A_even, float* d_A_odd)
{
    bool isOdd = true;
    int arr_length = ArraySize;
    int iterations = arr_length / BLOCK_SIZE + (arr_length % BLOCK_SIZE != 0 ? 1 : 0);
    while (arr_length != 1)
    {
        int GRID_SIZE = iterations / BLOCK_SIZE + (iterations % BLOCK_SIZE != 0 ? 1 : 0);
        if (isOdd)
        {
            sum_particles << <GRID_SIZE, BLOCK_SIZE >> > (d_A_even, arr_length, iterations);
            hipBindTexture(0, SumArrElementsRef, d_A_even, sizeof(float) * iterations);
        }
        else
        {
            sum_particles << <GRID_SIZE, BLOCK_SIZE >> > (d_A_odd, arr_length, iterations);
            hipBindTexture(0, SumArrElementsRef, d_A_odd, sizeof(float) * iterations);
        }
        hipDeviceSynchronize();

        arr_length = iterations;
        iterations = arr_length / BLOCK_SIZE + (arr_length % BLOCK_SIZE != 0 ? 1 : 0);
        isOdd = !isOdd;
    }
    float* result;
    result = (float*)malloc(sizeof(float));
    hipMemcpy(result, isOdd ? d_A_odd : d_A_even, sizeof(float), hipMemcpyDeviceToHost);
    return *result;
}

void generate_random_arr(float* A)
{
    for (int i = 0; i < ArraySize; i++)
    {
        A[i] = (rand() / (float)RAND_MAX) * (AbsMaxVal * 2 + 1) + -1 * AbsMaxVal;
        //printf("%f ", A[i]);
    }
    //printf("\n");
}