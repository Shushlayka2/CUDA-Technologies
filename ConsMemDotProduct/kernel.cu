#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <time.h>

using namespace std;

#define BLOCK_SIZE 256
#define ArraySize 1000
#define AbsMaxVal 10

void generate_random_arr(float* A);

__constant__ float A[ArraySize];
__constant__ float B[ArraySize];

__global__ void scalMult(float* C)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < ArraySize)
	{
		float sum = 0.0f;
		if (threadIdx.x == 0)
		{
			sum = 0.0;
			for (int j = 0; j < blockDim.x; j++)
				sum += A[j] * B[j];

			C[blockIdx.x] = sum;
		}
	}
}

int main()
{
	int GRID_SIZE = ArraySize / BLOCK_SIZE + (ArraySize % BLOCK_SIZE != 0 ? 1 : 0);
	srand(time(NULL));

	float *C, *Temp;
	float* d_C;
	C = (float*)malloc(sizeof(float) * GRID_SIZE);
	Temp = (float*)malloc(sizeof(float) * ArraySize);
	generate_random_arr(Temp);
	hipMemcpyToSymbol(HIP_SYMBOL(A), Temp, sizeof(float) * ArraySize);
	generate_random_arr(Temp);
	hipMemcpyToSymbol(HIP_SYMBOL(B), Temp, sizeof(float) * ArraySize);
	hipMalloc((void**)&d_C, sizeof(float) * GRID_SIZE);

	float KernelTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	scalMult << <GRID_SIZE, BLOCK_SIZE >> > (d_C);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&KernelTime, start, stop);

	hipMemcpy(C, d_C, sizeof(float) * GRID_SIZE, hipMemcpyDeviceToHost);
	float result = 0.0f;
	for (int i = 0; i < GRID_SIZE; i++)
		result += C[i];
	printf("Result: %f\nElapsedTime: %f", result, KernelTime);

	free(C);
	hipFree(d_C);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return 0;
}

void generate_random_arr(float* A)
{
	for (int i = 0; i < ArraySize; i++)
	{
		A[i] = (rand() / (float)RAND_MAX) * (AbsMaxVal * 2 + 1) + -1 * AbsMaxVal;
		printf("%f ", A[i]);
	}
	printf("\n");
}