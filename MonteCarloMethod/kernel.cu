#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

using namespace std;

#define N 409600
#define R 102400
#define SquaredR 10485760000

__global__ void calc(unsigned int* count, unsigned int seed)
{
	hiprandState_t state;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(seed, i, 0, &state);
	float x = hiprand_uniform(&state) * R;
	float y = hiprand_uniform(&state) * R;
	if (y * y <= SquaredR - x * x)
	{
		atomicInc(count, N);
	}
}

int main()
{
	unsigned int count;
	unsigned int *count_device;
	hipMalloc((void**)&count_device, sizeof(unsigned int));

	int blockSize = 32;
	int gridSize = N / blockSize + (N % blockSize == 0 ? 0 : 1);

	calc << <gridSize, blockSize >> > (count_device, time(NULL));
	hipDeviceSynchronize();
	hipMemcpy(&count, count_device, sizeof(unsigned int), hipMemcpyDeviceToHost);
	printf("%f", (count / (float)N) * 4.0f);
	hipFree(count_device);
	return 0;
}