#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <cmath> 
#include <stdio.h>
#include <malloc.h>

using namespace std;

#define ArrSize 20
#define BlockSize 32
#define BaseType double
#define DevOperation __exp10f//__expf//__dsqrt_rn//__fsqrt_rn
#define HostOperation(arg) exp10(arg)//exp//sqrt

#define STR_EXPAND(arg) #arg
#define STR(arg) STR_EXPAND(arg)

__global__ void calc(BaseType* A)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < ArrSize)
		A[i] = DevOperation(i);
}

int main()
{
	int GridSize = ArrSize / BlockSize + (ArrSize % BlockSize != 0 ? 1 : 0);
	size_t size = sizeof(BaseType) * ArrSize;

	BaseType* A_dev;
	hipMalloc((void**)&A_dev, size);

	float KernelTime;
	hipEvent_t start, stop;  
	hipEventCreate(&start);  
	hipEventCreate(&stop);

	hipEventRecord(start, 0);
	calc << <GridSize, BlockSize >> > (A_dev);
	hipEventRecord(stop, 0); 
	hipEventSynchronize(stop);
	hipEventElapsedTime(&KernelTime, start, stop);

	BaseType *A_host;
	A_host = (BaseType*)malloc(size);
	hipMemcpy(A_host, A_dev, size, hipMemcpyDeviceToHost);

	BaseType err = 0.0;
	for (int i = 0; i < ArrSize; i++)
	{
		printf("%.2f %.2f\n", HostOperation(i * 1.0), A_host[i]);
		err += abs(HostOperation(i * 1.0) - A_host[i]);
	}
		
	err /= ArrSize;

	printf("Used type is %s\nOperation is %s\nEllapsed time equals to %f milliseconds\nError equals to %f", 
		typeid(BaseType).name(), STR(DevOperation), KernelTime, err);

	hipFree(A_dev);
	hipEventDestroy(start);  
	hipEventDestroy(stop);
	return 0;
}